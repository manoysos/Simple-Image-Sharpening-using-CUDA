#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

struct PGMstruct
{
	int maxGrey;
	int width;
	int height;
	int **matrix;
	int **matrixTr;
};

typedef struct PGMstruct PGMImage;


__global__ void gpuSharpenImg3x3(int * d_matrix, int * d_matrixTr, int width, int height) //Kernel ��� ��� �������� ��� ������� 3x3 ��������������� ��� GPU.
{
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int upL = index - (width + 1);
	int upM = index - (width);
	int upR = index - (width - 1);
	int left = index - 1;
	int right = index + 1;
	int downL = index + (width - 1);
	int downM = index + (width);
	int downR = index + (width + 1);

	
	//����������� ���� ����� ���� ��� �� �������� ��� ��� ���������� ���� ������ ��� ���������� ������� ��� ������.
	if ((index > width - 1) && (index < (width*height) - width) && (index%width != 0) && (index%width != width - 1))
	{
		d_matrixTr[index] = (-1)*d_matrix[upL] + (-1)*d_matrix[upM] + (-1)*d_matrix[upR] + (-1)*d_matrix[left] + (-1)*d_matrix[right] + (-1)*d_matrix[downL] + (-1)*d_matrix[downM] + (-1)*d_matrix[downR] + (9 * d_matrix[index]);
		if (d_matrixTr[index]>255)
		{
			d_matrixTr[index] = 255;
		}
		if (d_matrixTr[index]<0)
		{
			d_matrixTr[index] = 0;
		}
	}  
	
	__syncthreads();
}


__global__ void gpuSharpenImg5x5(int*d_matrix, int*d_matrixTr, int width, int height)  //Kernel ��� ��� �������� ��� ������� 5x5 ��������������� ��� GPU.
{
	/*
	______________________________________
	
	|upUL  |upULM  |upUM  |upURM  |upUR  |
	|upL   |upLM   |upM   |upRM   |upR   |
	|leftL |left   |index |right  |rightR|
	|downL |downLM |downM |downRM |downR |
	|downDL|downDLM|downDM|downDRM|downDR|
	______________________________________
	
	*/
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int	upUL = index - 2 * width - 2;
	int	upULM = index - 2 * width - 1;
	int	upUM = index - 2 * width;
	int	upURM = index - 2 * width + 1;
	int	upUR = index - 2 * width + 2;
	int	upL = index - width - 2;
	int	upLM = index - width - 1;
	int	upM	 = index - width;
	int	upRM = index - width + 1;
	int	upR = index - width + 2;
	int	leftL = index - 2;
	int	left = index - 1;
	int	right = index + 1;
	int	rightR = index + 2;
	int	downL = index + width - 2;
	int	downLM = index + width - 1;
	int downM = index + width;
	int downRM = index + width + 1;
	int downR = index + width + 2;
	int downDL = index + 2 * width - 2;
	int downDLM = index + 2 * width - 1;
	int downDM = index + 2 * width;
	int downDRM = index + 2 * width + 1;
	int downDR = index + 2 * width + 2;

	//����������� ���� ����� ���� ��� �� �������� ��� ��� ���������� ���� ��� ������ ��� ��� ���������� ������� ��� ������.
	if ((index > 2 * width - 1) && (index < (width*height) - 2 * width) && (index%width != 0) && (index%width != width - 1) && (index%width != 1) && (index%width != width - 2))
	{
		d_matrixTr[index] = ((-1)*(d_matrix[upUL] + d_matrix[upULM] + d_matrix[upUM] + d_matrix[upURM] +
			d_matrix[upUR] + d_matrix[upL] + d_matrix[upR] + d_matrix[leftL] +
			d_matrix[rightR] + d_matrix[downL] + d_matrix[downR] + d_matrix[downDL] +
			d_matrix[downDLM] + d_matrix[downDM] + d_matrix[downDRM] + d_matrix[downDR])
			+ 2 * (d_matrix[upLM] + d_matrix[upM] + d_matrix[upRM] +
			d_matrix[left] + d_matrix[right] + d_matrix[downLM] +
			d_matrix[downM] + d_matrix[downRM]) + 8 * d_matrix[index]) / 8;
		if (d_matrixTr[index]>255)
		{
			d_matrixTr[index] = 255;
		}
		if (d_matrixTr[index]<0)
		{
			d_matrixTr[index] = 0;
		}
	}
	__syncthreads();
}

//��������� ��� ��� ������������ ��� ������ ��� ���� ����������� ��� ������� �������� ������ 2 ����������.
void deallocate_dynamic_matrix(int **matrix, int row)
{
	int i;

	for (i = 0; i < row; ++i)
		free(matrix[i]);
	free(matrix);
}

//��������� ��� ������� ��� �������� �� ������ pgm ��� ���� ���������� ��� ��� ������.
int getPGM(const char *flnm, PGMImage *pgm)
{
	FILE *pgmFile;
	char ch;
	int type, col, row;
	int ch_int;
	pgmFile = fopen(flnm, "rb");
	//������ ��������� �� ��� ������ �� �������� �� ������ ��� ���� ��������� � �������.
	if (!pgmFile)
	{
		perror("Cannot open file");

		exit(EXIT_FAILURE);
	}

	printf("\nReading image file: %s\n", pgmFile);

	ch = getc(pgmFile);
	if (ch != 'P')
	{
		printf("ERROR(1): Not valid pgm file type\n");
		exit(1);
	}
	ch = getc(pgmFile);
	/*��������� ��� char �� int ��� ��������� ��� ������� ��� ���� ��� �������*/
	type = ch - 48;
	if (type != 2)
	{
		printf("ERROR(2): Not valid pgm file type. Currently only P2 files can be used.\n");
		exit(1);
	}

	while (getc(pgmFile) != '\n');             // skip to end of line
	fseek(pgmFile, -1, SEEK_CUR);             // backup one character

	fscanf(pgmFile, "%d", &((*pgm).width));     //�������� ��� ���������� ��� ����� ��� ������� ��� �������
	fscanf(pgmFile, "%d", &((*pgm).height));    //�������� ��� ���������� ��� ����� ��� ����� ��� �������
	fscanf(pgmFile, "%d", &((*pgm).maxGrey));   //�������� ��� ���������� ��� ����� ��� �������� �������� ���� ��� �������

	printf("\n width  = %d", (*pgm).width);
	printf("\n height = %d", (*pgm).height);
	printf("\n maxVal = %d", (*pgm).maxGrey);
	printf("\n");

	if (type == 2) 
	{
		/*���������� ��� ��������� 2-��������� ������ "matrix". ���� ����� ������������ ����� ���� ������������ ������,
		������ ��� �� �� ���� ��� �������, � ������ �� �������� pointers ���� ������ �������������� �������, 
		������ ��� �� �� ������ ��� ������� � �������, ����� ������� ������ �� ������������� ��� ����� ��� ���� ���� pixel.*/
		
		pgm->matrix = (int **)malloc(sizeof(int *) * pgm->height);   //�������� ������ ��� ��� ������ �� �� pointers. 
		if (pgm->matrix == NULL) {                                   //�� ��������� ��������� ���� ��� �������� ������ ����������� ��������� ������.
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}

		for (int i = 0; i < pgm->height; i++) {
			(*pgm).matrix[i] = (int *)malloc(sizeof(int) * (*pgm).width); //�������� ������ ��� ���� ������ ��� �� ���������� ��� ����� ��� pixel 
			if ((*pgm).matrix[i] == NULL) {                               //��� ������� ��� pointer ���� ����� ���� ����������� ������.
				perror("memory allocation failure"); 					  //�� ��������� ��������� ���� ��� �������� ������ ����������� ��������� ������.
				exit(EXIT_FAILURE);
			}
		}

		for (row = 0; row <pgm->height; row++)
			for (col = 0; col< pgm->width; col++)
			{
				fscanf(pgmFile, "%d", &ch_int);						//�������� ��� ����� ��� ���� ��� ���� ����� ��� ���������� ��� ��� ���������� 
				(*pgm).matrix[row][col] = ch_int;					//�������� ��� ������������ ������ "matrix".
			}
	}
	fclose(pgmFile);
	return type;
}



void cpuSharpenImg3x3(PGMImage *pgm)					//��������� ��� ��� �������� ��� ������� 3x3 ��������������� ��� CPU.
{
	
	int adj = 0;
	int i, j, k, m;

	/*������������ ��� ���� ���� ����������� ������, ��� "matrixTr", ���� ����� �� ������������� 
	��� �������������� ������� �� �� ������ ����� ��� ������ "matrix".*/
	pgm->matrixTr = (int **)malloc(sizeof(int *) * pgm->height);
	if (pgm->matrixTr == NULL) {
		perror("memory allocation failure");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < pgm->height; i++) {
		pgm->matrixTr[i] = (int *)malloc(sizeof(int) * pgm->width);
		if (pgm->matrixTr[i] == NULL) {
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}
	}
	
	//���������� ���� ��� ��������� ��� ������ "matrix", ����� ��� ���� ��� ���������� ���� ������ ��� ���������� ������� ��� ������, 
	//�������� ��� ������� �� ���� ��� ���������� ���� ���� ������ "matrixTr".
	for (i = 1; i < pgm->height - 1; i++)
	{
		for (j = 1; j < pgm->width - 1; j++)
		{
			for (k = i - 1; k < i + 2; k++)
			{
				for (m = j - 1; m < j + 2; m++)
				{
					if (!((k == i) && (m == j)))
					{
						adj += pgm->matrix[k][m] * (-1);		
					}
				}
			}
			if ((pgm->matrix[i][j] * 9) + adj > 255)
			{
				pgm->matrixTr[i][j] = 255;
			}
			else if ((pgm->matrix[i][j] * 9) + adj <0)
			{
				pgm->matrixTr[i][j] = 0;
			}
			else
			{
				pgm->matrixTr[i][j] = ((pgm->matrix[i][j] * 9) + adj); 
			}
			adj = 0;
		}
	}
}

void cpuSharpenImg5x5(PGMImage *pgm)					//��������� ��� ��� �������� ��� ������� 5x5 ��������������� ��� CPU.
{
	int adj = 0;
	int i, j, k, m;
	/*������������ ��� ���� ���� ����������� ������, ��� "matrixTr", ���� ����� �� ������������� 
	��� �������������� ������� �� �� ������ ����� ��� ������ "matrix".*/
	pgm->matrixTr = (int **)malloc(sizeof(int *) * pgm->height);
	if (pgm->matrixTr == NULL) {
		perror("memory allocation failure");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < pgm->height; i++) {
		pgm->matrixTr[i] = (int *)malloc(sizeof(int) * pgm->width);
		if (pgm->matrixTr[i] == NULL) {
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}
	}

	//���������� ���� ��� ��������� ��� ������ "matrix", ����� ��� ���� ��� ���������� ���� 2 ������ ��� 2 ���������� ������� ��� ������, 
	//�������� ��� ������� �� ���� ��� ���������� ���� ���� ������ "matrixTr".
	for (i = 2; i < pgm->height - 2; i++)
	{
		for (j = 2; j < pgm->width - 2; j++)
		{
			for (k = i - 2; k < i + 3; k++)
			{
				for (m = j - 2; m < j + 3; m++)
				{
					if (!((k == i) && (m == j)))
					{
						if ((k == i - 2) || (k == i + 2) || (m == j - 2) || (m == j + 2))
						{
							adj += pgm->matrix[k][m] * (-1);
						}
						else
						{
							adj += pgm->matrix[k][m] * 2;
						}
					}

				}
			}
			if (((pgm->matrix[i][j] * 8) + adj) / 8 > 255)
			{
				pgm->matrixTr[i][j] = 255;
			}
			else if (((pgm->matrix[i][j] * 8) + adj) / 8 <0)
			{
				pgm->matrixTr[i][j] = 0;
			}
			else
			{
				pgm->matrixTr[i][j] = ((pgm->matrix[i][j] * 8) + adj) / 8;
			}
			adj = 0;
		}
	}
}

void crNewFile3x3(const char *flnm, PGMImage *pgm, int type)			//��������� ��� ��� ���������� ���������� ������� .pgm, �� ����� ��������� ��� ������������� �����, ������� �� �� ������ 3x3, ������.
{
	FILE *pgm3x3File;
	char *ending = "3x3.pgm\0";			//���������� ��� �� ����� ��� ����� ��� �������� ��� ���� �������.
	size_t len = strlen(flnm);
	size_t len2 = strlen(ending);
	char *finalName = (char*) malloc(len + len2);
	strcpy(finalName, flnm);			 //���������� ��� �������� ��� ���� ������� �� ��� ��������� ��� �������� ��� ������� �������
	strcat(finalName, ending);			 //��� ���� ���������� "3x3.pgm\0".
	int  i, j;

	pgm3x3File = fopen(finalName, "w");		//����������/������� ��� ������� ��� �������.
	
	//���������� ��� ����������� ��� �������(�����, ������, ���� ��� ������� ���� ����) ��� ������.
	fprintf(pgm3x3File, "%c", 'P');
	fprintf(pgm3x3File, "%d\n", 2);
	fprintf(pgm3x3File, "%d ", pgm->width);
	fprintf(pgm3x3File, "%d\n", pgm->height);
	fprintf(pgm3x3File, "%d\n", pgm->maxGrey);
	
	//���������� ��� ����� ��� ���� ���� ����� ��� ������.
	for (i = 0; i < pgm->height; i++)
	{
		for (j = 0; j < pgm->width; j++)
		{
			if ((i == 0) || (i == pgm->height - 1) || (j == 0) || (j == pgm->width - 1))
			{
				fprintf(pgm3x3File, "%d  ", pgm->matrix[i][j]);			//�������� ��� ����� ��� ��� ������ "matrix" ��� �� �������� ��� ��� ������ ��������� �� ������.

				if ((!((j == 0) && (i == 0))) && (j % 10 == 0))
				{
					fprintf(pgm3x3File, "%c\n", ' ');
				}
			}
			else
			{
				fprintf(pgm3x3File, "%d  ", pgm->matrixTr[i][j]);		//�������� ��� ����� ��� ��� ������ "matrixTr" ��� ��� �� �������� ��������.
				if (j % 10 == 0)
				{
					fprintf(pgm3x3File, "%c\n", ' ');
				}
			}
		}
	}
	fclose(pgm3x3File);
}

void crNewFile5x5(const char *flnm, PGMImage *pgm, int type)		//��������� ���������� ��� "crNewFile3x3", ���� ��� ������� ���� ������ ���� ���������� �� ������ 5x5.
{
	FILE *pgm5x5File;
	char *ending = "5x5.pgm\0";
	size_t len = strlen(flnm);
	size_t len2 = strlen(ending);
	char *finalName = (char*)malloc(len + len2);
	strcpy(finalName, flnm);
	strcat(finalName, ending);
	int  i, j;

	pgm5x5File = fopen(finalName, "w");
	fprintf(pgm5x5File, "%c", 'P');
	fprintf(pgm5x5File, "%d\n", 2);
	fprintf(pgm5x5File, "%d ", pgm->width);
	fprintf(pgm5x5File, "%d\n", pgm->height);
	fprintf(pgm5x5File, "%d\n", pgm->maxGrey);
	for (i = 0; i < pgm->height; i++)
	{
		for (j = 0; j < pgm->width; j++)
		{
			if ((i <2) || (i > pgm->height - 3) || (j <2) || (j > pgm->width - 3))
			{
				fprintf(pgm5x5File, "%d  ", pgm->matrix[i][j]);
				if ((j>0) && (j % 10 == 0))
				{
					fprintf(pgm5x5File, "%c\n", ' ');
				}
			}
			else
			{
				fprintf(pgm5x5File, "%d  ", pgm->matrixTr[i][j]);
				if (j % 10 == 0)
				{
					fprintf(pgm5x5File, "%c\n", ' ');
				}
			}
		}
	}
	fclose(pgm5x5File);
}

void createMTr(PGMImage *pgm)						//��������� ��� ��� �������� ������ ��� ��� ������ "matrixTr".
{
	pgm->matrixTr = (int **)malloc(sizeof(int *) * pgm->height);
	if (pgm->matrixTr == NULL) {
		perror("memory allocation failure");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < pgm->height; i++) {
		pgm->matrixTr[i] = (int *)malloc(sizeof(int) * pgm->width);
		if (pgm->matrixTr[i] == NULL) {
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}
	}
}

void populateHMatrix(PGMImage *pgm,int* h_matrix)			//��������� ��� ��� ��������� ���� ��������� ��� ������������ ������ "matrix" ���� ������������ ������ "h_matrix".
{
	int i, j;
	for (i = 0; i < pgm->height; i++)
	{
		for (j = 0; j < pgm->width; j++)
		{
			h_matrix[i*(pgm->height)+j] = pgm->matrix[i][j];
		}
	}
	
}

int main()
{
	clock_t t;
	clock_t startT;
	double dt;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int type;
	char choice;
	int * h_matrix; 
	int *  d_matrix;
	int * d_matrixTr;
	int m = 64; //������� threads.
	int n; //������� blocks.
	PGMImage img;
	FILE *logFile;
	startT = clock();
	t = startT;
	logFile = fopen("sharpenlog.txt", "w");//���������� ������� ��� ��� ��������� ��� ������ ��� ���������� �� ��������� ��� �� ����������� �������� ��������.
	printf("Please write the name of the file: \n");
	char filename[30];
	gets(filename);
	t = clock() - t;
	dt = ((double)t) / CLOCKS_PER_SEC;
	fprintf(logFile, "Filename was given after %f seconds.\n", dt);
	t = clock();
	type = getPGM(filename, &img);
	t = clock() - t;
	dt = ((double)t) / CLOCKS_PER_SEC;
	fprintf(logFile, "It took %f seconds to read the file.\n", dt);
	
	printf("Choose one of the following options by typing the appropriate number and pressing 'Enter': \n");
	printf("1. Apply 3x3 filter using the CPU.\n");
	printf("2. Apply 5x5 filter using the CPU.\n");
	printf("3. Apply 3x3 filter using the GPU.\n");
	printf("4. Apply 5x5 filter using the GPU.\n");
	choice = getchar();
	if (choice == '1')
	{
		t = clock();
		cpuSharpenImg3x3(&img); //�������� ��� ������� 3x3 �� ��� ����� CPU.
		t = clock() - t;
		dt = ((double)t) / CLOCKS_PER_SEC;
		fprintf(logFile, "It took %f seconds to apply the 3x3 filter.\n", dt);
		crNewFile3x3(filename, &img, type); //���������� ��� ���� ������� .pgm.
		deallocate_dynamic_matrix(img.matrixTr, img.height);
	}
	else if (choice == '2')
	{
		t = clock();
		cpuSharpenImg5x5(&img);  //�������� ��� ������� 5x5 �� ��� ����� CPU.
		t = clock() - t;
		dt = ((double)t) / CLOCKS_PER_SEC;
		fprintf(logFile, "It took %f seconds to apply the 5x5 filter.\n", dt);
		crNewFile5x5(filename, &img, type);  //���������� ��� ���� ������� .pgm.
		deallocate_dynamic_matrix(img.matrixTr, img.height);
	}
	else if (choice == '3')
	{
		n = (img.height*img.width) / m;//���������� ��� ������� ��� blocks.
		h_matrix = (int*)malloc(sizeof(int)*img.width*img.height); //�������� ������ ��� ��� ������ "h_matrix".
		if (h_matrix == NULL) 
		{
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}
		populateHMatrix(&img, h_matrix); //��������� ��� ��������� ��� "matrix" ���� "h_matrix".
		createMTr(&img);//�������� ������ ��� ��� ������ "matrixTr".
		t = clock();
		hipMalloc(&d_matrix, img.height*img.width*sizeof(int));//�������� ������ ���� GPU ��� ��� ������ "d_matrix".
		hipMalloc(&d_matrixTr, img.height*img.width*sizeof(int));//�������� ������ ���� GPU ��� ��� ������ "d_matrixTr".
		hipMemcpy(d_matrix, h_matrix, img.height*img.width*sizeof(int), hipMemcpyHostToDevice);//��������� ��� "h_matrix" ���� "d_matrix".
		hipDeviceSynchronize();
		t = clock() - t;
		dt = ((double)t) / CLOCKS_PER_SEC;
		fprintf(logFile, "It took %f seconds to allocate memory on the gpu and copy the data.\n", dt);
		
		hipEventRecord(start, 0);
		gpuSharpenImg3x3<<<n,m>>>(d_matrix,d_matrixTr, img.width, img.height);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipError_t errAsync = hipDeviceSynchronize();
		hipEventElapsedTime(&time, start, stop);
		fprintf(logFile, "It took %f ms to apply the 3x3 filter using the GPU.\n", time);
		hipMemcpy(h_matrix, d_matrixTr, img.height*img.width*sizeof(int), hipMemcpyDeviceToHost);//��������� ��� "d_matrixTr" ���� "h_matrix".
		hipDeviceSynchronize();
		for (int i = 0; i < img.height; i++)
		{
			for (int j = 0; j < img.width; j++)
			{
				img.matrixTr[i][j] = h_matrix[i*(img.height/* - 1*/) + j];//��������� ��� ��������� ��� "h_matrix" ���� "matrixTr".
			}
		}  
		crNewFile3x3(filename, &img, type);	   //������� ��� ���������� ��� ��� ���������� ��� ���� �������.
		
		//�������� ������� ��� ������������ ��� ������ ��� ���� ��������� ��� ������.
		hipFree(d_matrix);
		hipFree(d_matrixTr);
		hipDeviceSynchronize();
		deallocate_dynamic_matrix(img.matrixTr, img.height);
		free(h_matrix);			  
	}
	else if (choice == '4')
	{
		n = (img.height*img.width) / m;//���������� ��� ������� ��� blocks.
		h_matrix = (int*)malloc(sizeof(int)*img.width*img.height); //�������� ������ ��� ��� ������ "h_matrix".
		if (h_matrix == NULL) {
			perror("memory allocation failure");
			exit(EXIT_FAILURE);
		}
		populateHMatrix(&img, h_matrix);  //��������� ��� ��������� ��� "matrix" ���� "h_matrix".
		createMTr(&img);  //�������� ������ ��� ��� ������ "matrixTr".
		t = clock();
		hipMalloc(&d_matrix, img.height*img.width*sizeof(int));//�������� ������ ���� GPU ��� ��� ������ "d_matrix".
		hipMalloc(&d_matrixTr, img.height*img.width*sizeof(int));//�������� ������ ���� GPU ��� ��� ������ "d_matrixTr".
		hipMemcpy(d_matrix, h_matrix, img.height*img.width*sizeof(int), hipMemcpyHostToDevice);//��������� ��� "h_matrix" ���� "d_matrix".
		hipDeviceSynchronize();
		t = clock() - t;
		dt = ((double)t) / CLOCKS_PER_SEC;
		fprintf(logFile, "It took %f seconds to allocate memory on the gpu and copy the data.\n", dt);
		hipEventRecord(start, 0);
		gpuSharpenImg5x5<<<n,m>>>(d_matrix,d_matrixTr, img.width, img.height);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipError_t errAsync = hipDeviceSynchronize();
		
		hipEventElapsedTime(&time, start, stop);
		fprintf(logFile, "It took %f ms to apply the 5x5 filter using the GPU.\n", time);
		hipMemcpy(h_matrix, d_matrixTr, img.height*img.width*sizeof(int), hipMemcpyDeviceToHost);//��������� ��� "d_matrixTr" ���� "h_matrix".
		hipDeviceSynchronize();
		for (int i = 0; i < img.height; i++)
		{
			for (int j = 0; j < img.width; j++)
			{
				img.matrixTr[i][j] = h_matrix[i*(img.height/* - 1*/) + j];//��������� ��� ��������� ��� "h_matrix" ���� "matrixTr".
			}
		}  
		crNewFile5x5(filename, &img, type);		   //������� ��� ���������� ��� ��� ���������� ��� ���� �������.

		//�������� ������� ��� ������������ ��� ������ ��� ���� ��������� ��� ������.
		hipFree(d_matrix);
		hipFree(d_matrixTr);
		hipDeviceSynchronize();
		deallocate_dynamic_matrix(img.matrixTr, img.height);
		free(h_matrix);
	}
	else
	{
		printf("Invalid option.");
		deallocate_dynamic_matrix(img.matrix, img.height);
		exit(1);
	}
	deallocate_dynamic_matrix(img.matrix, img.height);
	t = clock() - startT;
	dt = ((double)t) / CLOCKS_PER_SEC;
	fprintf(logFile, "It took %f seconds for the program to complete.\n", dt);
	fclose(logFile);
	return 0;

}